// #include <cub/device/device_reduce.cuh>
#include <cub/device/device_reduce.cuh>
#include <stdio.h>

// CustomMin functor
struct CustomMin
{
	template <typename T>
	HIPCUB_RUNTIME_FUNCTION __forceinline__ __device__
	T operator()(const T &a, const T &b) const {
		return (b < a) ? b : a;
	}
};

using namespace hipcub;


void print_int_array(int* a, int n) {
	printf("[ ");
	for (int i = 0; i < n; i++)
		printf("%d ", a[i]);
	printf("]\n");
}

int main() {
	// Declare, allocate, and initialize device-accessible pointers for input and output
	int          num_items = 8;        // e.g., 8
	int          *d_keys_in;         // e.g., [0, 2, 2, 9, 5, 5, 5, 8]
	int          *d_values_in;       // e.g., [0, 7, 1, 6, 2, 5, 3, 4]
	int          *d_unique_out;      // e.g., [-, -, -, -, -, -, -, -]
	int          *d_aggregates_out;  // e.g., [-, -, -, -, -, -, -, -]
	int          *d_num_runs_out;    // e.g., [-]
	CustomMin    reduction_op;

	hipMallocManaged((void**)&d_keys_in, sizeof(int) * num_items);
	hipMallocManaged((void**)&d_values_in, sizeof(int) * num_items);
	hipMallocManaged((void**)&d_unique_out, sizeof(int) * num_items);
	hipMallocManaged((void**)&d_aggregates_out, sizeof(int) * num_items);
	hipMallocManaged((void**)&d_num_runs_out, sizeof(int));
	int i = 0;
	d_keys_in[i++] = 0; d_keys_in[i++] = 2; d_keys_in[i++] = 2; d_keys_in[i++] = 9;
	d_keys_in[i++] = 5; d_keys_in[i++] = 5; d_keys_in[i++] = 5; d_keys_in[i++] = 8;

	i = 0;
	d_values_in[i++] = 0; d_values_in[i++] = 7; d_values_in[i++] = 1; d_values_in[i++] = 6;
	d_values_in[i++] = 2; d_values_in[i++] = 5; d_values_in[i++] = 3; d_values_in[i++] = 4;

	print_int_array(d_keys_in, 8);
	print_int_array(d_values_in, 8);

	// Determine temporary device storage requirements
	void     *d_temp_storage = NULL;
	size_t   temp_storage_bytes = 1024;
	hipcub::DeviceReduce::ReduceByKey(d_temp_storage, temp_storage_bytes, d_keys_in, d_unique_out, d_values_in, d_aggregates_out, d_num_runs_out, reduction_op, num_items);
	// Allocate temporary storage
	hipMalloc(&d_temp_storage, temp_storage_bytes);
	// Run reduce-by-key
	hipcub::DeviceReduce::ReduceByKey(d_temp_storage, temp_storage_bytes, d_keys_in, d_unique_out, d_values_in, d_aggregates_out, d_num_runs_out, reduction_op, num_items);
	// d_unique_out      <-- [0, 2, 9, 5, 8]
	// d_aggregates_out  <-- [0, 1, 6, 2, 4]
	// d_num_runs_out    <-- [5]

	printf("d_num_runs_out %d",d_num_runs_out[0]);
	print_int_array(d_unique_out, d_num_runs_out[0]);
	print_int_array(d_aggregates_out, d_num_runs_out[0]);
	printf("success\n");
	return 0;
}